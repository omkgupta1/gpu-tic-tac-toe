#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "common.h"

__global__ void gpuChooseMove1(char *board, int *move) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    hiprand_init(1234, idx, 0, &state);

    int attempt = 0;
    do {
        attempt = hiprand(&state) % BOARD_SIZE;
    } while (!isValidMove(board, attempt));

    *move = attempt;
}
