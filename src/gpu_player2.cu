#include "hip/hip_runtime.h"
#include "common.h"

__global__ void gpuChooseMove2(char *board, int *move) {
    // Simple strategy: pick the first valid move
    for (int i = 0; i < BOARD_SIZE; ++i) {
        if (isValidMove(board, i)) {
            *move = i;
            return;
        }
    }
    *move = -1;
}
